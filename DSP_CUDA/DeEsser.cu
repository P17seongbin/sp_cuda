#include "hip/hip_runtime.h"
#include "CUDAlib.cuh"
#include "WAVFilter.h"
#include <chrono>
#define BLOCK_SIZE 64

__global__ void FilterCompressorBlock(char* d_in, char* d_out, double threshold, double ratio)
{
	__shared__ int s_data[BLOCK_SIZE];
	int d_data;

	int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
	int src_x = dst_x + offset;
	// Load one element per thread from device memory and store it 

	d_data = d_in[dst_x];
	if (dst_x >= offset)
		s_data[threadIdx.x] = d_in[src_x];
	else
		s_data[threadIdx.x] = 0;

	d_data += s_data[threadIdx.x] * gain;
	d_out[dst_x] = d_data;
}

void FilterCompressor(Audio_WAV& origin, bool useCUDA, double threshold, double ratio)
{
	WAV_HEADER origin_header = origin.get_header();
	size_t memSize = origin_header.Subchunk2Size;
	char* origin_bytes = origin.get_audio();

	if (useCUDA)
	{
		//pointer for device
		char* d_in, * d_out;

		int numBlocks = (memSize / BLOCK_SIZE) + 1; //celling
		int sharedMemSize = BLOCK_SIZE; //one byte for each thread

		hipMalloc((void**)& d_in, memSize);
		hipMalloc((void**)& d_out, memSize);

		hipMemcpy(d_in, origin_bytes, memSize, hipMemcpyHostToDevice);
		// launch kernel
		dim3 dimGrid(numBlocks);
		dim3 dimBlock(BLOCK_SIZE);
		FilterDeEsserBlock << < dimGrid, dimBlock, sharedMemSize >> > (d_in, d_out, threshold, ratio);

		hipDeviceSynchronize();

		hipMemcpy(origin_bytes, d_out, memSize, hipMemcpyDeviceToHost);

		hipFree(d_in);
		hipFree(d_out);
	}
	else
	{
		char* origin_archive = new char[memSize];
		std::memcpy((char*)origin_archive, (char*)origin_bytes, memSize);

		for (int i = offset + 1; i < memSize; i++)
		{
			origin_bytes[i] = origin_archive[i] + origin_archive[i - offset] * gain;
		}
	}
}
