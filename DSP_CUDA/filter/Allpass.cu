#include "hip/hip_runtime.h"
#include "CUDAlib.cuh"
#include "WAVFilter.h"
#include <chrono>
#define BLOCK_SIZE 64

__global__ void FilterAllPassBlock(char* d_in, char* d_out, int offset, double gain)
{
	__shared__ int s_data[BLOCK_SIZE];
	int d_data;

	int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
	int src_x = dst_x + offset;
	double t1 = (1 - (gain * gain));
	// Load one element per thread from device memory and store it 

	d_data = d_in[dst_x];
	if (dst_x >= offset)
		s_data[threadIdx.x] = d_in[src_x];
	else
		s_data[threadIdx.x] = 0;

	d_data += s_data[threadIdx.x] * gain;

	d_out[dst_x] = t1 * d_data - gain * s_data[threadIdx.x];
}
/*allpass came from 
* https://medium.com/the-seekers-project/coding-a-basic-reverb-algorithm-part-2-an-introduction-to-audio-programming-4db79dd4e325
* Written by Rishikesh Daoo 
*/

//RT for Reverbation Time
void FilterAllpass(Audio_WAV& origin, bool useCUDA, double delay, double gain, double RT) 
{
	WAV_HEADER origin_header = origin.get_header();
	size_t memSize = origin_header.Subchunk2Size;
	char* origin_bytes = origin.get_audio();
	int offset = origin_header.sampleRate * delay;

	if (useCUDA)
	{
		//pointer for device
		char* d_in, * d_out;

		int numBlocks = (memSize / BLOCK_SIZE) + 1; //celling
		int sharedMemSize = BLOCK_SIZE; //one byte for each thread

		hipMalloc((void**)& d_in, memSize);
		hipMalloc((void**)& d_out, memSize);

		hipMemcpy(d_in, origin_bytes, memSize, hipMemcpyHostToDevice);
		// launch kernel
		dim3 dimGrid(numBlocks);
		dim3 dimBlock(BLOCK_SIZE);
		FilterAllPassBlock << < dimGrid, dimBlock, sharedMemSize >> > (d_in, d_out, offset, gain);

		hipDeviceSynchronize();

		hipMemcpy(origin_bytes, d_out, memSize, hipMemcpyDeviceToHost);

		hipFree(d_in);
		hipFree(d_out);
	}
	else
	{
		char* delayed_sound = new char[memSize];
		double t1 = (1 - (gain * gain));
		std::memcpy((char*)delayed_sound, (char*)origin_bytes, memSize);

		for (int i = offset + 1; i < memSize; i++)
		{
			delayed_sound[i] += delayed_sound[i - offset] * gain;
		}
		for (int i = 0; i < memSize; i++)
		{
			origin_bytes[i] = -1 * gain * origin_bytes[i] + t1 * delayed_sound[i];
		}
	}
}
