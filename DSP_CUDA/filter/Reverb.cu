#include "CUDAlib.cuh"
#include "WAVFilter.h"
#include <chrono>
#define BLOCK_SIZE 64

/*allpass came from 
* https://medium.com/the-seekers-project/coding-a-basic-reverb-algorithm-part-2-an-introduction-to-audio-programming-4db79dd4e325
* Written by Rishikesh Daoo 
*/

//RT for Reverbation Time
void FilterAllpass(Audio_WAV& origin, bool useCUDA, double delay, double gain, double RT) 
{
	WAV_HEADER origin_header = origin.get_header();
	size_t memSize = origin_header.Subchunk2Size;
	char* origin_bytes = origin.get_audio();
	int offset = origin_header.sampleRate * delay;

	if (useCUDA)
	{
		//pointer for device
		char* d_in, * d_out;

		int numBlocks = (memSize / BLOCK_SIZE) + 1; //celling
		int sharedMemSize = BLOCK_SIZE; //one byte for each thread

		hipMalloc((void**)& d_in, memSize);
		hipMalloc((void**)& d_out, memSize);

		hipMemcpy(d_in, origin_bytes, memSize, hipMemcpyHostToDevice);
		// launch kernel
		dim3 dimGrid(numBlocks);
		dim3 dimBlock(BLOCK_SIZE);
		FilterEchoBlock << < dimGrid, dimBlock, sharedMemSize >> > (d_in, d_out, offset, gain);

		hipDeviceSynchronize();

		hipMemcpy(origin_bytes, d_out, memSize, hipMemcpyDeviceToHost);

		hipFree(d_in);
		hipFree(d_out);
	}
	else
	{
		char* delayed_sound = new char[memSize];
		std::memcpy((char*)delayed_sound, (char*)origin_bytes, memSize);

		for (int i = 0; i < memSize; i++)
		{
			if(i > offset)
				delayed_sound[i] =  delayed_sound[i - offset] * gain;
		}

		
	}
}
