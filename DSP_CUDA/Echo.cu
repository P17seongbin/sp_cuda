#include "hip/hip_runtime.h"
#include "CUDAlib.cuh"
#include "WAVFilter.h"
#include <chrono>
#define BLOCK_SIZE 64

__global__ void FilterEchoBlock(char* d_in, char* d_out, int offset)
{
	__shared__ int s_data[BLOCK_SIZE];
	int d_data;

	int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
	int src_x = dst_x + offset;
	// Load one element per thread from device memory and store it 
	// *in reversed order* into temporary shared memory

	d_data = d_in[dst_x];
	if (dst_x >= offset)
		s_data[threadIdx.x] = d_in[src_x];
	else
		s_data[threadIdx.x] = 0;

	// Block until all threads in the block have written their data to shared mem

	__syncthreads();

	d_data += s_data[threadIdx.x];
	d_out[dst_x] = d_data;
}

void FilterEchoCUDA(Audio_WAV& origin)
{
	WAV_HEADER origin_header = origin.get_header();
	size_t memSize  = origin_header.Subchunk2Size;
	char* origin_bytes = origin.get_audio();
	int offset = origin_header.sampleRate / 4;

	//pointer for device
	char *d_in, * d_out;

	int numBlocks = (memSize / BLOCK_SIZE) + 1; //celling
	int sharedMemSize = BLOCK_SIZE; //one byte for each thread

	hipMalloc((void**) &d_in, memSize);
	hipMalloc((void**) &d_out, memSize);

	hipMemcpy(d_in, origin_bytes, memSize, hipMemcpyHostToDevice);
	// launch kernel
	dim3 dimGrid(numBlocks);
	dim3 dimBlock(BLOCK_SIZE);
	FilterEchoBlock << < dimGrid, dimBlock, sharedMemSize >> > (d_in, d_out, offset);


	hipDeviceSynchronize();

	hipMemcpy(origin_bytes, d_out, memSize, hipMemcpyDeviceToHost);


	hipFree(d_in);
	hipFree(d_out);

}
