#include "hip/hip_runtime.h"
#include "CUDAlib.cuh"
#include "WAVFilter.h"
#include <chrono>
#define BLOCK_SIZE 64

// reduce bit value to half means -6dB

__global__ void FilterCompressorBlock(char* d_in, char* d_out, double threshold, double ratio, int bitperSample)
{
	double max_dB = 6.02 * bitperSample; // approx. value of SNR. https://en.wikipedia.org/wiki/Audio_bit_depth#Bit_rate_and_file_size

	int byterate = bitperSample / 8;
	short d_data = 0;
	int x = (blockIdx.x * blockDim.x + threadIdx.x) * byterate;
	// Load one element per thread from device memory and store it 

	memcpy(&d_data, (d_in + x), byterate);

	double dB;

	if (d_data < 0)
		dB = max_dB + 10 * log10((double)(-1 * d_data)) - 3.012 * bitperSample;
	else
		dB = max_dB + 10 * log10((double)(d_data)) - 3.012 * bitperSample;

	if (dB > threshold)
	{
		dB = ratio * (dB - threshold) + threshold;
		dB += (3.012 * bitperSample - max_dB);
		dB /= 10;
		short v = (short)pow(10.0, dB);
		d_data = (d_data < 0) ? (-1 * v) : v;
	}

	memcpy((d_out + x), &d_data, byterate);
	
}

void FilterCompressor(Audio_WAV& origin, bool useCUDA, double threshold, double ratio)
{
	WAV_HEADER origin_header = origin.get_header();
	size_t memSize = origin_header.Subchunk2Size;
	char* origin_bytes = origin.get_audio();
	int bitperSample = origin_header.bitsPerSample;
	int byterate = bitperSample / 8;

	if (useCUDA)
	{
		//pointer for device
		char* d_in, * d_out;

		int numBlocks = (memSize / BLOCK_SIZE) + 1; //celling
		int sharedMemSize = BLOCK_SIZE * byterate; //number byte for each thread

		hipMalloc((void**)& d_in, memSize);
		hipMalloc((void**)& d_out, memSize);

		hipMemcpy(d_in, origin_bytes, memSize, hipMemcpyHostToDevice);
		// launch kernel
		dim3 dimGrid(numBlocks);
		dim3 dimBlock(BLOCK_SIZE / byterate);

		FilterCompressorBlock << < dimGrid, dimBlock, sharedMemSize >> > (d_in, d_out, threshold, ratio, bitperSample);

		hipDeviceSynchronize();

		memset(origin_bytes, 0, memSize);

		char* origin_archive = new char[memSize];
		hipMemcpy(origin_archive, d_out, memSize, hipMemcpyDeviceToHost);

		origin.set_audio(origin_archive);

		hipFree(d_in);
		hipFree(d_out);
	}
	else
	{

		double max_dB = 6.02 * bitperSample; // approx. value of SNR. https://en.wikipedia.org/wiki/Audio_bit_depth#Bit_rate_and_file_size
		for (int i = 0; i < memSize; i += byterate)
		{
			short value;
			memcpy(&value, &origin_bytes[i], byterate);
			double dB;

			if (value < 0)
				dB = max_dB + 10 * log10((double)(-1 * value)) - 3.012 * bitperSample;
			else
				dB = max_dB + 10 * log10((double)(value)) - 3.012 * bitperSample;

			if (dB > threshold)
			{
				dB = ratio * (dB - threshold) + threshold;
				dB += (3.012 * bitperSample - max_dB);
				dB /= 10;
				short v = (short)pow(10, dB);
				value = (value < 0) ? (-1 * v) : v;
			}
			memcpy(&origin_bytes[i], &value, byterate);
		}
	}
}
