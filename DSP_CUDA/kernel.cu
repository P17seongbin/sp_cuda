#include "CUDAlib.cuh"
#include "Filter.cuh"
#include "FileHandler.h"

#include <string>
#include <iostream>

/*
argument list:
-echo x y : apply echo effect x times with delay of y seconds
*/
int main(int argc, char* argv[])
{
	//�ϴ��� argument���� ����
	FILE *audiofile = NULL;
	audio_type filetype;

	std::string filename; 

	//Manual input mode
	if (argc < 3)
	{
		std::cout << "Type audio file path (WAV, MPC, FLAC) : ";
		std::cin >> filename;	
	}
	else
	{
		filename = argv[1];
	}
	if ((audiofile = open_audio(filename, &filetype)) == NULL)
	{ 
		std::cout << "Something went wrong while opening file \"" << filename << "\"" << std::endl;
		std::cout << "Aborting...." << std::endl;
	}
	
}