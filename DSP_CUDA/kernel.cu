#include "CUDAlib.cuh"
#include "Filter.cuh"
#include "FileHandler.h"

#include <string>
#include <iostream>
#include <map>

/*
argument list:
-echo x y : apply echo effect x times with delay of y seconds
*/
int main(int argc, char* argv[])
{
	static std::map<std::string, FileOpenerType> opener_map;
	if (!opener_created)
	{
		opener_map.insert(make_pair("wav", open_wav));
		opener_created = true;
	}
	//�ϴ��� argument���� ����
	FILE *audiofile = NULL;
	audio_type filetype;

	std::string filename; 

	//Manual input mode
	if (argc < 3)
	{
		std::cout << "Type audio file path (WAV, MPC, FLAC) : ";
		std::cin >> filename;	
	}
	else
	{
		filename = argv[1];
	}

	//Open file
	if ((audiofile = fopen(filename.c_str(), "r")) == NULL)
	{	
		std::cout << "Something went wrong while opening file \"" << filename << "\"" << std::endl;
		std::cout << "Aborting...." << std::endl;
	}
	
	//after opening file, determine given file's type 
	//do something 
	//�ϴ��� wav�� �Ѵٰ� ����


}